#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits>
#include <vector>
#include <cmath>
#include <chrono>

#define NUM_HORMIGAS 5
#define NUM_CIUDADES 5
#define ALPHA 1.0f
#define BETA 5.0f
#define EVAPORACION 0.5f
#define FEROMONA_INICIAL 1.0f
#define ITERACIONES 100
#define COSTO_PENALIZADO 100000.0f

__device__ float distancias[NUM_CIUDADES][NUM_CIUDADES] = {
    {0, 3, 9, 2, 5}, 
    {3, 0, 4, 8, 2}, 
    {9, 4, 0, 8, 1}, 
    {2, 8, 8, 0, 4}, 
    {5, 2, 1, 4, 0}  
};

__device__ float feromonas[NUM_CIUDADES][NUM_CIUDADES];

__global__ void inicializarFeromonas() {
    int i = threadIdx.x;
    int j = threadIdx.y;
    if (i < NUM_CIUDADES && j < NUM_CIUDADES) {
        feromonas[i][j] = FEROMONA_INICIAL;
    }
}

__device__ float calcularProbabilidad(int ciudad_actual, int ciudad_siguiente, bool* visitado) {
    if (visitado[ciudad_siguiente] || distancias[ciudad_actual][ciudad_siguiente] == 0) return 0.0f;
    return powf(feromonas[ciudad_actual][ciudad_siguiente], ALPHA) *
           powf(1.0f / distancias[ciudad_actual][ciudad_siguiente], BETA);
}

__global__ void construirSoluciones(int* rutas, float* costos, hiprandState* states, float* probabilidades_host) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= NUM_HORMIGAS) return;

    bool visitado[NUM_CIUDADES] = {false};
    int ruta[NUM_CIUDADES];
    float costo = 0.0f;

    hiprandState localState = states[id];
    int ciudad_actual = hiprand(&localState) % NUM_CIUDADES;
    ruta[0] = ciudad_actual;
    visitado[ciudad_actual] = true;

    for (int step = 1; step < NUM_CIUDADES; ++step) {
        float total_probabilidad = 0.0f;

        for (int j = 0; j < NUM_CIUDADES; ++j) {
            float prob = calcularProbabilidad(ciudad_actual, j, visitado);
            total_probabilidad += prob;
            probabilidades_host[id * NUM_CIUDADES + j] = prob;
        }

        float prob_seleccion = hiprand_uniform(&localState) * total_probabilidad;
        float suma_probabilidad = 0.0f;
        int siguiente_ciudad = -1;
        for (int j = 0; j < NUM_CIUDADES; ++j) {
            suma_probabilidad += probabilidades_host[id * NUM_CIUDADES + j];
            if (suma_probabilidad >= prob_seleccion) {
                siguiente_ciudad = j;
                break;
            }
        }

        ruta[step] = siguiente_ciudad;
        visitado[siguiente_ciudad] = true;
        costo += distancias[ciudad_actual][siguiente_ciudad];
        ciudad_actual = siguiente_ciudad;
    }
    costo += distancias[ciudad_actual][ruta[0]];

    if (costo == 0) {
        costo = COSTO_PENALIZADO;
    }

    for (int i = 0; i < NUM_CIUDADES; ++i) {
        rutas[id * NUM_CIUDADES + i] = ruta[i];
    }
    costos[id] = costo;
    states[id] = localState;
}

__global__ void actualizarFeromonas(int* rutas, float* costos) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    if (i < NUM_CIUDADES && j < NUM_CIUDADES) {
        feromonas[i][j] *= (1.0f - EVAPORACION);
    }

    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < NUM_HORMIGAS) {
        int* ruta = &rutas[id * NUM_CIUDADES];
        float costo = costos[id];
        if (costo != COSTO_PENALIZADO) {
            for (int k = 0; k < NUM_CIUDADES - 1; ++k) {
                int a = ruta[k];
                int b = ruta[k + 1];
                atomicAdd(&feromonas[a][b], 1.0f / costo);
                atomicAdd(&feromonas[b][a], 1.0f / costo);
            }
        }
    }
}

__global__ void inicializarEstados(hiprandState* states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &states[id]);
}

void imprimirInformacion(int* rutas, float* costos, float* probabilidades, int num_hormigas) {
    for (int i = 0; i < num_hormigas; ++i) {
        std::cout << "Hormiga " << i << ": Ruta = ";
        for (int j = 0; j < NUM_CIUDADES; ++j) {
            std::cout << rutas[i * NUM_CIUDADES + j] << " ";
        }
        std::cout << "| Costo = " << costos[i];

        std::cout << "| Probabilidades: ";
        for (int j = 0; j < NUM_CIUDADES; ++j) {
            std::cout << probabilidades[i * NUM_CIUDADES + j] << " ";
        }
        std::cout << std::endl;
    }
}


int main() {
    int* d_rutas;
    float* d_costos;
    hiprandState* d_states;
    float* d_probabilidades;

    hipMalloc(&d_rutas, NUM_HORMIGAS * NUM_CIUDADES * sizeof(int));
    hipMalloc(&d_costos, NUM_HORMIGAS * sizeof(float));
    hipMalloc(&d_states, NUM_HORMIGAS * sizeof(hiprandState));
    hipMalloc(&d_probabilidades, NUM_HORMIGAS * NUM_CIUDADES * sizeof(float));

    inicializarFeromonas<<<1, dim3(NUM_CIUDADES, NUM_CIUDADES)>>>();
    inicializarEstados<<<NUM_HORMIGAS / 32 + 1, 32>>>(d_states);

    int* h_rutas = new int[NUM_HORMIGAS * NUM_CIUDADES];
    float* h_costos = new float[NUM_HORMIGAS];
    float* h_probabilidades = new float[NUM_HORMIGAS * NUM_CIUDADES];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int iter = 0; iter < ITERACIONES; ++iter) {
        construirSoluciones<<<NUM_HORMIGAS / 32 + 1, 32>>>(d_rutas, d_costos, d_states, d_probabilidades);
        actualizarFeromonas<<<NUM_HORMIGAS / 32 + 1, 32>>>(d_rutas, d_costos);

        hipMemcpy(h_rutas, d_rutas, NUM_HORMIGAS * NUM_CIUDADES * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_costos, d_costos, NUM_HORMIGAS * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_probabilidades, d_probabilidades, NUM_HORMIGAS * NUM_CIUDADES * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "Iteración " << iter + 1 << std::endl;
        imprimirInformacion(h_rutas, h_costos, h_probabilidades, NUM_HORMIGAS);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Tiempo total de ejecución: " << elapsedTime << " ms" << std::endl;

    hipFree(d_rutas);
    hipFree(d_costos);
    hipFree(d_states);
    hipFree(d_probabilidades);
    delete[] h_rutas;
    delete[] h_costos;
    delete[] h_probabilidades;

    std::cout << "Algoritmo ACO en CUDA completado." << std::endl;
    return 0;
}